#include "hip/hip_runtime.h"
#include "add.h"

__global__ void add_kernel(const float *a, const float *b, float *c, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

torch::Tensor add(const torch::Tensor &a, const torch::Tensor &b) {

    torch::Tensor c = torch::empty_like(a);
    int n = a.numel();
    int threads = 512;
    int blocks = (n + threads - 1) / threads;

    add_kernel<<<blocks, threads>>>(
        a.data_ptr<float>(), 
        b.data_ptr<float>(), 
        c.data_ptr<float>(), 
        n);

    return c;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("add", &add, "Add two tensors (CUDA)");
}